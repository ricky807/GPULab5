
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 4 // Number of threads in each thread block
 
/* 
 * CUDA kernel to find a global max, each thread process
 * one element.
 * @param values	input of an array of integers in which we search a max number
 * @param max		output of this kernel, the max number in array values 
 * @param reg_maxes	output of this kernel, some regional max number for input array
 * @param num_regions	input of this kernel, number of regions we use to reduce lock contentions
 * @param n		input of this kernel, total number of element in input array
 */
__global__ void global_max(int *values, int *max, int *reg_maxes, int num_regions, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x; 
    int val = values[i];
    int region = i % num_regions; 
    if(atomicMax(&reg_maxes[region],val) < val) 
    { 
        atomicMax(max,val); 
    }
}

// Write the cuda kernel to normal all elements in input values,
// the output is stored back into output array, max is the maximum value in the array
// values, n is the total number of elements in values.
__global__ void normalize(int *values, int *max, float *output, int n)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x; 
    if(i < n)
    {
        output[i] = (float) values[i] / (float)max[0];
    }

}   
 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int i;
    int input[] = {4, 5, 6, 7, 19, 10, 0, 4, 2, 3, 1, 7, 9, 11, 45, 23, 100, 29};
    int n = sizeof(input) / sizeof(float); //careful, this usage only works with statically allocated arrays, NOT dynamic arrays

    // Host input vectors
    int *h_in = input;
    //Host output vector
    float *h_out = (float *) malloc(n * sizeof(float));
 
    // Device input vectors
    int *d_in;;

    //Device output vector
    float *d_out;
    int *d_reg_max;// memory for regional max
    int *d_gl_max;  // memory for global max
 
    // Size, in bytes, of each vector
    int bytes = n * sizeof(int);
    int num_reg = ceil(n / (float)BLOCKSIZE);  //num of regions we will use in calculation of global max
   
    // Allocate memory for each vector on GPU
    hipMalloc(&d_in, bytes);
    hipMalloc(&d_out, n * sizeof(float));
    hipMalloc(&d_reg_max, num_reg * sizeof(int) );
    hipMalloc(&d_gl_max, sizeof(int) );
 
    //PLEASE initialize the values in d_reg_max and d_gl_max to ZERO!!!


 
    // Copy host data to device
    hipMemcpy( d_in, h_in, bytes, hipMemcpyHostToDevice);
 
    // Number of threads in each thread block
    int blockSize = BLOCKSIZE;
 
    // Number of thread blocks in grid
    int gridSize = (int)ceil((float)n/blockSize);
 
    //printf("BlockSize: %d, Gridsize: %d", blockSize, gridSize);
    // Execute the kernel
    global_max<<<gridSize, blockSize>>>(d_in, d_gl_max, d_reg_max, num_reg, n); //after this kernel called, *d_gl_max is ready to use
    hipDeviceSynchronize();
 
    // Execute the second kernel, use the data returned by the first kernel
    normalize<<<gridSize, blockSize>>>(d_in, d_gl_max, d_out, n); 
 
    // Copy array back to host
    hipMemcpy( h_out, d_out, n * sizeof(float), hipMemcpyDeviceToHost );
 
    // Show the result
    printf("The original array is: ");
    for(i = 0; i < n; i ++)
        printf("%6d,", h_in[i] );    
    
    printf("\n\nNormalized   array is: ");
    for(i = 0; i < n; i++)
        printf("%6.2f,", h_out[i] );    
    puts("");
    
    // Release device memory
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_reg_max);
    hipFree(d_gl_max);
 
    // Release host memory
    free(h_out);
 
    return 0;
}
